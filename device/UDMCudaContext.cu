//
// Created by dbj on 24-5-3.
//

#include "UDMCudaContext.cuh"

#include "hip/hip_runtime.h"

__host__ void cu_calloc_dev(void *ptr_d, int aligned_size, uint64_t nelem, uint64_t size) {
    uint64_t total_length = (nelem / aligned_size + 1) * aligned_size * size;
    hipError_t err = hipMalloc(&ptr_d, total_length);
};